#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <torch/library.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/Dispatch.h>

#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/threadblock/mma_pipelined.h>
#include <cutlass/gemm/threadblock/default_mma_core.h>
#include <cutlass/cutlass.h>
#include <cute/tensor.hpp>
#include "extended_gemm.h"
#include "extended_gemm_collective_api.h"

namespace at {
namespace native {

void _extended_gemm_kernel(float * a_ptr, float * b_ptr, float * out_ptr, int M, int N, int K, int lda, int ldb, int ldc) {
  // Option 1: 创建 CUDA 流
  // hipStream_t stream;
  // hipStreamCreate(&stream);

  // Option 2: Use the CUDA Stream from PyTorch
  hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  using Gemm = cutlass::gemm::device::Gemm<
    float, cutlass::layout::RowMajor,
    float, cutlass::layout::RowMajor,
    float, cutlass::layout::RowMajor
  >;

  Gemm gemm_op;
  float alpha = 1.0;
  float beta = 0.0;
  Gemm::Arguments args(
    {M, N, K},
    {a_ptr, lda},
    {b_ptr, ldb},
    {out_ptr, ldc},
    {out_ptr, ldc},
    {alpha, beta});
  cutlass::Status status = gemm_op(args);

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "CUTLASS GEMM 计算失败!" << std::endl;
  }

  // // 同步 CUDA Stream
  // hipStreamSynchronize(stream);
  // hipStreamDestroy(stream);
}

__global__ void _extended_gemm_block_naive_kernel(float * a_ptr, float * b_ptr, float * out_ptr, int M, int N, int K, int lda, int ldb, int ldc) {
    // naive cuda implementation, each block calculate a output element
    auto m = blockIdx.x;
    auto n = threadIdx.x;
    if (m < M && n < N) {
      out_ptr[m * ldc + n] = 0.0;
      for (int k=0; k<K; k++) {
        out_ptr[m * ldc + n] += a_ptr[m * lda + k] * b_ptr[k * ldb + n];
      }
    }
}

template <typename T, typename T2, int kTileM, int kTileN, int kTileK, typename TiledMMA, bool use_relu, bool use_slm, std::enable_if_t<!use_slm, int> =0>
__global__ void _extended_gemm_block_cutlass_naive_kernel(
  T * a_ptr,
  T * b_ptr,
  T2 * out_ptr,
  int M,
  int N,
  int K,
  int lda,
  int ldb,
  int ldc) {
    // 构造CUTE Tensor, size 是总的Tensor    
    cute::Tensor A = cute::make_tensor(cute::make_gmem_ptr(a_ptr), cute::make_shape(M, K), cute::make_stride(K, cute::Int<1>{}));
    cute::Tensor B = cute::make_tensor(cute::make_gmem_ptr(b_ptr), cute::make_shape(N, K), cute::make_stride(K, cute::Int<1>{}));  // Column Major
    cute::Tensor C = cute::make_tensor(cute::make_gmem_ptr(out_ptr), cute::make_shape(M, N), cute::make_stride(N, cute::Int<1>{}));

    // 当前block 线程组 要处理的 Tensor Tile 
    int ix = blockIdx.x;  // N 维度，因为 定义 grid(grid_n, grid_m) 
    int iy = blockIdx.y;  // M 维度

    // gA(kTileM, kTileK, num_tile_k)
    // gB(kTileN, kTileK, num_tile_k)
    // gC(kTileM, kTileN) 
    cute::Tensor gA = cute::local_tile(A, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileK>{}), cute::make_coord(iy, cute::_));
    cute::Tensor gB = cute::local_tile(B, cute::make_tile(cute::Int<kTileN>{}, cute::Int<kTileK>{}), cute::make_coord(ix, cute::_));
    cute::Tensor gC = cute::local_tile(C, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileN>{}), cute::make_coord(iy, ix));

    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);

    // MMA 表示 tiled_mma 单条指令要用到的元素个数
    //   1. refer to: https://github.com/NVIDIA/cutlass/blob/5e497243f7ad13a2aa842143f9b10bbb23d98292/media/docs/cpp/cute/0x_gemm_tutorial.md#tiledmma
    //   2. https://github.com/NVIDIA/cutlass/blob/5e497243f7ad13a2aa842143f9b10bbb23d98292/media/docs/cpp/cute/0t_mma_atom.md#type-aliases
    //      就是 汇编代码 这里的 A，B, C, D 寄存器的大小 对应的元素的个数
    //      以 SM80_16x8x16_F32F16F16F32_TN 为例子：https://github.com/NVIDIA/cutlass/blob/main/include/cute/arch/mma_sm80.hpp
    //      print(tAgA) 看到 MMA 是(2, 2, 2) 表示 这个线程 会处理 8 = 16*16/32 (M*K/num_thread_of_warp)个 A 的元素，
    //        对应了汇编代码中的 a0-a3 x 32bit 寄存器 共 8 * fp16
    //      print(tBgB) 看到 MMA 是(2, 2) 表示 这个线程 会处理 4 = 8*16/32 (N*K/num_thread_of_warp)个 B 的元素，
    //        对应了汇编代码中的 b0-b1 x 32bit 寄存器 共 4 * fp16
    //      print(tCgC) 看到 MMA 是(2, 2) 表示 这个线程 会处理 4 = 8*16/32 (N*K/num_thread_of_warp)个 C 的元素，
    //        对应了汇编代码中的 c0-c3 x 32bit 寄存器 共 4 * fp32
    //      A, B, C 矩阵 每个线程具体操作的元素排布: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#warp-level-matrix-fragment-mma-16816-float
    //      * tAgA gmem_ptr[16b](0x7f5ff3208000) o ((_2,_2,_2),_1,_2,4):((_1,1024,_8),_0,_16,_32)
    //        * 先看 size
    //          * 首先 (_2,_2,_2) 每个线程 分到 8个元素，4 组 每组 2个 {a0, a1}, {a2, a3}, {a4, a5}, {a6, a7}
    //          * 再看 _1, kTileM (32) 需要 1个 tiled_mma (32)
    //          * 再看 _2, kTileK (32) 需要 2个 tiled_mma (16)
    //          * 再看 4 (注意这个4是动态的，和真实K大小相关), 整个K 是128， 需要4个 kTileK
    //        * 再看 stride
    //          * (_1,1024,_8)
    //             * _1 因为 a0 a1 的间隔是 1
    //             * 1024 （注意是动态的 和K相关），a0 和 a2 差了 8行，每行K(128) 个元素，1024 = 8 * K （128）
    //             * _8, a0 和 a4 之间的间隔是8
    //          * _0, MMA_M 的size 就是1，不需要stride
    //          * _16, MMA_K 两个 tiled_mma 之间 沿着 K 维度 间隔了 16个元素
    //          * _32, 两个 kTileK 之间 沿着 K 维度间隔了 32个元素
    // MMA_M, MMA_K 表示 kTileM, kTileK 按照 tiled_mma 划分需要计算的次数
    // num_tile_k 表示一共有多少个 kTileK
    auto tAgA = thr_mma.partition_A(gA);  // (MMA, MMA_M, MMA_K, num_tile_k)
    auto tBgB = thr_mma.partition_B(gB);  // (MMA, MMA_N, MMA_K, num_tile_k)
    auto tCgC = thr_mma.partition_C(gC);  // (MMA, MMA_M, MMA_N)

    // if (cute::thread0()) {
    //   printf("\n");
    //   print(tAgA);
    //   printf("\n");
    //   print(tBgB);
    //   printf("\n");
    //   print(tCgC);
    //   printf("\n");
    // }

    // 返回寄存器声明
    auto tArA = thr_mma.partition_fragment_A(gA(cute::_, cute::_, 0));  // (MMA, MMA_M, MMA_K)
    auto tBrB = thr_mma.partition_fragment_B(gB(cute::_, cute::_, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrC = thr_mma.partition_fragment_C(gC(cute::_, cute::_));     // (MMA, MMA_M, MMA_N)
  
    // set to zero
    cute::clear(tCrC);

    int num_tile_k = cute::size<2>(gA);
    #pragma unroll 1
    for(int itile = 0; itile < num_tile_k; ++itile) {
      cute::copy(tAgA(cute::_, cute::_, cute::_, itile), tArA);
      cute::copy(tBgB(cute::_, cute::_, cute::_, itile), tBrB);

      cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
    }

    if (use_relu) {
      // 手动遍历并应用 ReLU
      // cute::size(tCrC) 是 8 = （kTileM * kTileN） / threadIdx.x
      // 所以应该就表示了 当前 threadIdx.x 要处理的元素
      CUTE_UNROLL
      for (int i = 0; i < cute::size(tCrC); ++i) {
          T2 val = tCrC(i);  // 取出当前值
          tCrC(i) = (val > T2(0)) ? val : T2(0);  // 应用 ReLU 操作
      }

      __syncthreads();
    }

    cute::copy(tCrC, tCgC);
}


template <typename T, typename T2, int kTileM, int kTileN, int kTileK, typename TiledMMA, bool use_relu, bool use_slm, std::enable_if_t<use_slm, int> =0>
__global__ void _extended_gemm_block_cutlass_naive_kernel(
  T * a_ptr,
  T * b_ptr,
  T2 * out_ptr,
  int M,
  int N,
  int K,
  int lda,
  int ldb,
  int ldc) {
    // This version will use share local memory 
    // 当前block 线程组 要处理的 Tensor Tile 
    int ix = blockIdx.x;  // N 维度，因为 定义 grid(grid_n, grid_m) 
    int iy = blockIdx.y;  // M 维度

    // Copy data from HBM to SLM
    extern __shared__ cute::half_t shared_array[];

    // // Option 1: Navie Copy from HBM to SLM
    // auto num_thread = blockDim.x;
    // int num_element_to_copy_per_thread = kTileM * K / num_thread;
    // int start_idx = threadIdx.x * num_element_to_copy_per_thread;
    // for (int i = 0; i < num_element_to_copy_per_thread ; i++) {
    //   shared_array[start_idx + i] = *(a_ptr + iy * kTileM * K + start_idx + i);
    // }

    // Option 2: Use tiledCopy
    // **重要** 这里我们希望 TiledCopy 每次 copy kTileM * kTileK 大小的块, 循环 K / kTileK 次
    // SM80_CP_ASYNC_CACHEGLOBAL 的介绍: https://zhuanlan.zhihu.com/p/1904236341904009066
    // Example Code: https://github.com/reed-lau/cute-gemm/blob/51dc19e783cd4b722177a6b5637a03db2d2851a9/gemm-multi-stage.cu#L94
    constexpr int kNThreads = size(TiledMMA{}); // In typical case, we use 128 threads
    constexpr int kNWarps = kNThreads / 32; // In typical case, we use 4 warp
    using Gmem_copy_struct = cute::SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;  // 一个 Atom 要copy的 bits 就是这里给的 cute::uint128_t
    // 因为 我们定义的 copy ATOM 是 cute::SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>
    // kGmemElemsPerLoad 表示 一个 ATOM copy 8 个 fp16 = 128 个bit
    static constexpr int kGmemElemsPerLoad = sizeof(cute::uint128_t) / sizeof(cute::half_t);
    
    // TiledCopy 每次 copy kTileM * kTileK 大小的块
    // 一个 ATOM 沿着 K 维度 copy 8个元素，所以需要4个线程沿着 K 维度 完成 copy
    static constexpr int kNThreadsK = kTileK / kGmemElemsPerLoad;
    // 一共 128 线程，128 // kNThreadsK = kNThreadsM = 32
    // 需要 32 个线程 沿着 M 维度完成Copy
    static constexpr int kNThreadsM = kNThreads / kNThreadsK;
    using GmemLayoutAtom = cute::Layout<cute::Shape<cute::Int<kNThreadsM>, cute::Int<kNThreadsK>>, cute::Stride<cute::Int<kNThreadsK>, cute::_1>>;  // (kTileM, 4)

    // Thr layout: ThrLayout 表示如何从执行线程的层面对 单个Copy_Atom进行扩展，所有值相乘 等于 线程的数量
    // Val layout: 这里的 val layout 必须是8的整数倍，8 从哪里来的: 因为 cute::SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>，一次copy 8 个 half
    //    如果是8的倍数，应该就表示了这个copy atom 要循环 copy 多次
    using GmemTiledCopyQKV = decltype(
        make_tiled_copy(cute::Copy_Atom<Gmem_copy_struct, cute::half_t>{},
                        GmemLayoutAtom{},  // Thr Layout
                        cute::Layout<cute::Shape<cute::_1, cute::Int<kGmemElemsPerLoad>>>{})); // Val layout
    GmemTiledCopyQKV gmem_tiled_copy_QKV;
    auto gmem_thr_copy_QKV = gmem_tiled_copy_QKV.get_thread_slice(threadIdx.x);


    cute::Tensor mgA = cute::make_tensor(cute::make_gmem_ptr(a_ptr), cute::make_shape(M, K), cute::make_stride(K, cute::Int<1>{}));
    cute::Tensor gA = cute::local_tile(mgA, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileK>{}), cute::make_coord(iy, cute::_)); // gA(kTileM, kTileK, num_tile_k)
    cute::Tensor msA = cute::make_tensor(cute::make_smem_ptr(shared_array), cute::make_shape(kTileM, K), cute::make_stride(K, cute::Int<1>{}));
    cute::Tensor sA_new = cute::local_tile(msA, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileK>{}), cute::make_coord(0, cute::_));

    // CPY_M 表示 
    // CPY_K 表示 
    // k 表示 整个 K 沿着 kTileK 需要循环多少次 来copy
    cute::Tensor tAgA_new = gmem_thr_copy_QKV.partition_S(gA);  // (CPY, CPY_M, CPY_K, k)
    cute::Tensor tAsA_new = gmem_thr_copy_QKV.partition_D(sA_new); // (CPY, CPY_M, CPY_K, k)


    if (cute::thread0()) {
      printf("\n");
      print(tAgA_new);  // gmem_ptr[16b](0x7f6923208000) o ((_8,_1),_1,_1,4):((_1,_0),_0,_0,_32)
      printf("\n");
      print(tAsA_new);  // smem_ptr[16b](0x7f6945000000) o ((_8,_1),_1,_1,4):((_1,_0),_0,_0,_32)
      printf("\n");
    }

    #pragma unroll 1
    for(int itile = 0; itile < cute::size<2>(gA); ++itile) {
      // 这里 我们每次 循环是 拷贝 kTileM * kTileK 大小的块
      // 一共循环 K / kTileK 次
      cute::copy(gmem_tiled_copy_QKV, tAgA_new(cute::_, cute::_, cute::_, itile), tAsA_new(cute::_, cute::_, cute::_, itile));
    }
    cute::cp_async_fence();
    // 这里我们完全阻塞了，等待所有的数据从 HBM 向 SLM copy 完成
    // 优化的写法，可以等一块 itile copy 完了，再去async copy 下一块，同时进行这一块的计算
    // 对于 cp_async_fence 以及 _cp_async_wait的解释，参考: https://zhuanlan.zhihu.com/p/1904236341904009066
    cute::cp_async_wait<0>();
    __syncthreads();


    // 构造CUTE Tensor, size 是总的Tensor       
    cute::Tensor A = cute::make_tensor(cute::make_smem_ptr(shared_array), cute::make_shape(kTileM, K), cute::make_stride(K, cute::Int<1>{}));
    cute::Tensor B = cute::make_tensor(cute::make_gmem_ptr(b_ptr), cute::make_shape(N, K), cute::make_stride(K, cute::Int<1>{}));  // Column Major
    cute::Tensor C = cute::make_tensor(cute::make_gmem_ptr(out_ptr), cute::make_shape(M, N), cute::make_stride(N, cute::Int<1>{}));

    // gA(kTileM, kTileK, num_tile_k)
    // gB(kTileN, kTileK, num_tile_k)
    // gC(kTileM, kTileN) 
    cute::Tensor sA = cute::local_tile(A, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileK>{}), cute::make_coord(0, cute::_));
    cute::Tensor gB = cute::local_tile(B, cute::make_tile(cute::Int<kTileN>{}, cute::Int<kTileK>{}), cute::make_coord(ix, cute::_));
    cute::Tensor gC = cute::local_tile(C, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileN>{}), cute::make_coord(iy, ix));

    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
  
    auto tAsA = thr_mma.partition_A(sA);  // (MMA, MMA_M, MMA_K, num_tile_k)
    auto tBgB = thr_mma.partition_B(gB);  // (MMA, MMA_N, MMA_K, num_tile_k)
    auto tCgC = thr_mma.partition_C(gC);  // (MMA, MMA_M, MMA_N)

    // 返回寄存器声明
    auto tArA = thr_mma.partition_fragment_A(sA(cute::_, cute::_, 0));  // (MMA, MMA_M, MMA_K)
    auto tBrB = thr_mma.partition_fragment_B(gB(cute::_, cute::_, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrC = thr_mma.partition_fragment_C(gC(cute::_, cute::_));     // (MMA, MMA_M, MMA_N)
  
    // set to zero
    cute::clear(tCrC);

    // if (cute::thread0()) {
    //   printf("\n");
    //   print(tAsA);
    //   printf("\n");
    //   print(tArA);
    //   printf("\n");
    // }

    // TODO<leslie> using tiled copy from smem to register
    using SmemCopyAtom = cute::Copy_Atom<cute::SM75_U32x4_LDSM_N, cute::half_t>;
    auto smem_tiled_copy_A = make_tiled_copy_A(SmemCopyAtom{}, tiled_mma);
    auto smem_thr_copy_A = smem_tiled_copy_A.get_thread_slice(threadIdx.x);

    cute::Tensor tArA_copy_view = smem_thr_copy_A.retile_D(tArA);

    auto tAsA_copy = smem_thr_copy_A.partition_S(sA);

    // if (cute::thread0()) {
    //   printf("\n");
    //   print(tAsA_copy);
    //   printf("\n");
    //   print(tArA_copy_view);
    //   printf("\n");
    // }

    int num_tile_k = cute::size<2>(sA);
    #pragma unroll 1
    for(int itile = 0; itile < num_tile_k; ++itile) {
      cute::copy(smem_tiled_copy_A, tAsA_copy(cute::_, cute::_, cute::_, itile), tArA_copy_view);
      cute::copy(tBgB(cute::_, cute::_, cute::_, itile), tBrB);
      cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
    }

    if (use_relu) {
      // 手动遍历并应用 ReLU
      // cute::size(tCrC) 是 8 = （kTileM * kTileN） / threadIdx.x
      // 所以应该就表示了 当前 threadIdx.x 要处理的元素
      CUTE_UNROLL
      for (int i = 0; i < cute::size(tCrC); ++i) {
          T2 val = tCrC(i);  // 取出当前值
          tCrC(i) = (val > T2(0)) ? val : T2(0);  // 应用 ReLU 操作
      }

      __syncthreads();
    }

    cute::copy(tCrC, tCgC);
}

template <typename input_dtype, typename output_dtype, bool use_relu, std::enable_if_t<!std::is_same_v<input_dtype, Half>, int> =0>
void _extended_gemm_kernel_low_level_api(
  input_dtype * a_ptr,
  input_dtype * b_ptr,
  output_dtype * out_ptr,
  int M,
  int N,
  int K,
  int lda,
  int ldb,
  int ldc) {
  TORCH_CHECK(false, "None Half input not support yet");
}


template <typename input_dtype, typename output_dtype, bool use_relu, std::enable_if_t<std::is_same_v<input_dtype, Half>, int> =0> // std::enable_if_t<std::is_same_v<input_dtype, Half>, int> =0
void _extended_gemm_kernel_low_level_api(
  input_dtype * a_ptr,
  input_dtype * b_ptr,
  output_dtype * out_ptr,
  int M,
  int N,
  int K,
  int lda,
  int ldb,
  int ldc) {
  constexpr int kTileM = 32;
  constexpr int kTileN = 32;
  constexpr int kTileK = 32;
  int grid_m = M / kTileM;
  int grid_n = N / kTileN;

  using mma_op = cute::SM80_16x8x16_F16F16F16F16_TN;
  using mma_traits = cute::MMA_Traits<mma_op>;
  using mma_atom = cute::MMA_Atom<mma_traits>;

  // mma_op will calculate MNK as 16x8x16，tensor core 作用在warp上，所以是一个warp的32线程 的计算size
  // 创建 tiled mma 通过 make_layout 可以进一步扩展  tiled_mma 计算size
  // param: thr layout 表示可以有更多的 warp(线程 来参与 mma计算)， 这里就是 M 维度是2个，N 维度是2个
  // param: val layout (**重要** 最新的code 下面 val layout 的参数被移除了) 表示 在 warp 内部，可以重复参与的mma 计算，这里表示 沿着 N 维度重复2次
  // param: 最后一个参数是 Permutations 被保留了： 
  //   refer to: https://github.com/NVIDIA/cutlass/blob/5e497243f7ad13a2aa842143f9b10bbb23d98292/include/cute/atom/mma_atom.hpp#L207
  //   refer to: https://zhuanlan.zhihu.com/p/28168438940
  // 综上：
  // 一共有 4 个 warp，128个线程参与计算, size(MMA{}) 返回的也是这个结构 (32, cute::_2, cute::_2, cute::_1)
  using MMA = decltype(make_tiled_mma(mma_atom{}, 
                      make_layout(cute::Shape<cute::_2, cute::_2, cute::_1>{}))); // thr layout
                      // make_layout(cute::Shape<cute::_1, cute::_2, cute::_1>{}))); // val layout has been removed


  using mma_op_fp32 = cute::SM80_16x8x16_F32F16F16F32_TN;
  using mma_traits_fp32 = cute::MMA_Traits<mma_op_fp32>;
  using mma_atom_fp32 = cute::MMA_Atom<mma_traits_fp32>;
  using MMA_fp32 = decltype(make_tiled_mma(mma_atom_fp32{}, 
                      make_layout(cute::Shape<cute::_2, cute::_2, cute::_1>{}), // thr layout
                      cute::Tile<cute::_32, cute::_16, cute::_16>{})); // permutation

  dim3 grid(grid_n, grid_m);
  dim3 block;
  using T = cute::half_t;
  using T2 = float;
  if constexpr (std::is_same_v<output_dtype, at::Half>) {
    block = dim3(size(MMA{}));
    _extended_gemm_block_cutlass_naive_kernel<T, T, kTileM, kTileN, kTileK, MMA, use_relu, false><<<grid, block>>>(
      reinterpret_cast<T*>(a_ptr), reinterpret_cast<T*>(b_ptr), reinterpret_cast<T*>(out_ptr), M, N, K, lda, ldb, ldc
    );
  } else {
    block = dim3(size(MMA_fp32{}));
    
    bool use_slm = true;
    if (!use_slm) {
      _extended_gemm_block_cutlass_naive_kernel<T, T2, kTileM, kTileN, kTileK, MMA_fp32, use_relu, false><<<grid, block>>>(
        reinterpret_cast<T*>(a_ptr), reinterpret_cast<T*>(b_ptr), (T2*)out_ptr, M, N, K, lda, ldb, ldc
      );
    } else {
      size_t shared_mem_size_in_bytes = kTileM * K * sizeof(cute::half_t);
      _extended_gemm_block_cutlass_naive_kernel<T, T2, kTileM, kTileN, kTileK, MMA_fp32, use_relu, true><<<grid, block, shared_mem_size_in_bytes>>>(
        reinterpret_cast<T*>(a_ptr), reinterpret_cast<T*>(b_ptr), (T2*)out_ptr, M, N, K, lda, ldb, ldc
      ); 
    }
  }

}

Tensor extended_gemm_kernel(
  Tensor a,
  Tensor b,
  Tensor out,
  std::string_view epilogue,
  bool transpose_B,
  int64_t api_level) {
    if (epilogue == "none" && !transpose_B && api_level == 0) {
      // High level API not support epilogue
      // TODO<leslie> assert the scalar_type is float, and the input tensor is 2D
      auto a_ptr = a.data_ptr();
      auto b_ptr = b.data_ptr();
      auto out_ptr = out.data_ptr();

      int M = a.size(0);
      int K = a.size(1);
      int N = b.size(1);

      int lda = a.size(1);
      int ldb = b.size(1);
      int ldc = out.size(1);

      _extended_gemm_kernel((float*)a_ptr, (float*)b_ptr, (float*)out_ptr, M, N, K, lda, ldb, ldc);
    } else if (api_level == 1) {
      // Collective API
      int M = a.size(0);
      int K = a.size(1);
      int N = b.size(0);

      int lda = a.size(1);
      int ldb = b.size(1);
      int ldc = out.size(1);
      // std::cout<<std::is_same_v<c10::impl::ScalarTypeToCPPTypeT<at::ScalarType::Half>, Half><<std::endl;
      AT_DISPATCH_FLOATING_TYPES_AND2(
          at::ScalarType::BFloat16, at::ScalarType::Half, out.scalar_type(),
          "_extended_gemm_kernel_collective_api_kernel_impl",
          [&] { 
            // std::cout<<std::is_same_v<scalar_t, Half><<std::endl;
            Half* a_ptr = a.data_ptr<Half>();
            Half* b_ptr = b.data_ptr<Half>();
            scalar_t* out_ptr = out.data_ptr<scalar_t>();
            if (epilogue == "relu") {
              _extended_gemm_kernel_collective_api<Half, scalar_t, true>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            } else {
              _extended_gemm_kernel_collective_api<Half, scalar_t, false>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            }
          });
    
    }
    else {
      // A is: M x K
      // B is: N x K
      // C is: M x N
      TORCH_CHECK(transpose_B, "for cute api, transpose_B must be true");

      int M = a.size(0);
      int K = a.size(1);
      int N = b.size(0);

      int lda = a.size(1);
      int ldb = b.size(1);
      int ldc = out.size(1);
      // std::cout<<std::is_same_v<c10::impl::ScalarTypeToCPPTypeT<at::ScalarType::Half>, Half><<std::endl;
      AT_DISPATCH_FLOATING_TYPES_AND2(
          at::ScalarType::BFloat16, at::ScalarType::Half, out.scalar_type(),
          "_extended_gemm_kernel_low_level_api_kernel_impl",
          [&] { 
            // std::cout<<std::is_same_v<scalar_t, Half><<std::endl;
            at::Half* a_ptr = a.data_ptr<at::Half>();
            at::Half* b_ptr = b.data_ptr<at::Half>();
            scalar_t* out_ptr = out.data_ptr<scalar_t>();
            if (epilogue == "relu") {
              _extended_gemm_kernel_low_level_api<at::Half, scalar_t, true>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            } else {
              _extended_gemm_kernel_low_level_api<at::Half, scalar_t, false>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            }
          });
    }

    return out;
}

} // namespace native
} // namespace at
