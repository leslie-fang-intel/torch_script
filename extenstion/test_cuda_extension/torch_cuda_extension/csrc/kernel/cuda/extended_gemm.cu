#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <torch/library.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/Dispatch.h>

#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/threadblock/mma_pipelined.h>
#include <cutlass/gemm/threadblock/default_mma_core.h>
#include <cutlass/cutlass.h>
#include <cute/tensor.hpp>
#include "extended_gemm.h"
#include "extended_gemm_collective_api.h"

namespace at {
namespace native {

void _extended_gemm_kernel(float * a_ptr, float * b_ptr, float * out_ptr, int M, int N, int K, int lda, int ldb, int ldc) {
  // Option 1: 创建 CUDA 流
  // hipStream_t stream;
  // hipStreamCreate(&stream);

  // Option 2: Use the CUDA Stream from PyTorch
  hipStream_t stream = c10::cuda::getCurrentCUDAStream();

  using Gemm = cutlass::gemm::device::Gemm<
    float, cutlass::layout::RowMajor,
    float, cutlass::layout::RowMajor,
    float, cutlass::layout::RowMajor
  >;

  Gemm gemm_op;
  float alpha = 1.0;
  float beta = 0.0;
  Gemm::Arguments args(
    {M, N, K},
    {a_ptr, lda},
    {b_ptr, ldb},
    {out_ptr, ldc},
    {out_ptr, ldc},
    {alpha, beta});
  cutlass::Status status = gemm_op(args);

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "CUTLASS GEMM 计算失败!" << std::endl;
  }

  // // 同步 CUDA Stream
  // hipStreamSynchronize(stream);
  // hipStreamDestroy(stream);
}

__global__ void _extended_gemm_block_naive_kernel(float * a_ptr, float * b_ptr, float * out_ptr, int M, int N, int K, int lda, int ldb, int ldc) {
    // naive cuda implementation, each block calculate a output element
    auto m = blockIdx.x;
    auto n = threadIdx.x;
    if (m < M && n < N) {
      out_ptr[m * ldc + n] = 0.0;
      for (int k=0; k<K; k++) {
        out_ptr[m * ldc + n] += a_ptr[m * lda + k] * b_ptr[k * ldb + n];
      }
    }
}

template <typename T, typename T2, int kTileM, int kTileN, int kTileK, typename TiledMMA, bool use_relu>
__global__ void _extended_gemm_block_cutlass_naive_kernel(
  T * a_ptr,
  T * b_ptr,
  T2 * out_ptr,
  int M,
  int N,
  int K,
  int lda,
  int ldb,
  int ldc) {
    // 构造CUTE Tensor, size 是总的Tensor    
    cute::Tensor A = cute::make_tensor(cute::make_gmem_ptr(a_ptr), cute::make_shape(M, K), cute::make_stride(K, cute::Int<1>{}));
    cute::Tensor B = cute::make_tensor(cute::make_gmem_ptr(b_ptr), cute::make_shape(N, K), cute::make_stride(K, cute::Int<1>{}));  // Column Major
    cute::Tensor C = cute::make_tensor(cute::make_gmem_ptr(out_ptr), cute::make_shape(M, N), cute::make_stride(N, cute::Int<1>{}));

    // 当前block 线程组 要处理的 Tensor Tile 
    int ix = blockIdx.x;  // N 维度，因为 定义 grid(grid_n, grid_m) 
    int iy = blockIdx.y;  // M 维度

    // gA(kTileM, kTileK, num_tile_k)
    // gB(kTileN, kTileK, num_tile_k)
    // gC(kTileM, kTileN) 
    cute::Tensor gA = cute::local_tile(A, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileK>{}), cute::make_coord(iy, cute::_));
    cute::Tensor gB = cute::local_tile(B, cute::make_tile(cute::Int<kTileN>{}, cute::Int<kTileK>{}), cute::make_coord(ix, cute::_));
    cute::Tensor gC = cute::local_tile(C, cute::make_tile(cute::Int<kTileM>{}, cute::Int<kTileN>{}), cute::make_coord(iy, ix));

    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);

    // MMA 表示 tiled_mma 单条指令要用到的元素个数
    //   1. refer to: https://github.com/NVIDIA/cutlass/blob/5e497243f7ad13a2aa842143f9b10bbb23d98292/media/docs/cpp/cute/0x_gemm_tutorial.md#tiledmma
    //   2. https://github.com/NVIDIA/cutlass/blob/5e497243f7ad13a2aa842143f9b10bbb23d98292/media/docs/cpp/cute/0t_mma_atom.md#type-aliases
    //      感觉就是 这里的 A，B, C, D 寄存器的大小 对应的元素的个数
    // MMA_M, MMA_K 表示 kTileM, kTileK 按照 tiled_mma 划分需要计算的次数
    // num_tile_k 表示一共有多少个 kTileK
    auto tAgA = thr_mma.partition_A(gA);  // (MMA, MMA_M, MMA_K, num_tile_k)
    auto tBgB = thr_mma.partition_B(gB);  // (MMA, MMA_N, MMA_K, num_tile_k)
    auto tCgC = thr_mma.partition_C(gC);  // (MMA, MMA_M, MMA_N)

    // 返回寄存器声明
    auto tArA = thr_mma.partition_fragment_A(gA(cute::_, cute::_, 0));  // (MMA, MMA_M, MMA_K)
    auto tBrB = thr_mma.partition_fragment_B(gB(cute::_, cute::_, 0));  // (MMA, MMA_N, MMA_K)
    auto tCrC = thr_mma.partition_fragment_C(gC(cute::_, cute::_));     // (MMA, MMA_M, MMA_N)
  
    // set to zero
    cute::clear(tCrC);

    int num_tile_k = cute::size<2>(gA);
    #pragma unroll 1
    for(int itile = 0; itile < num_tile_k; ++itile) {
      cute::copy(tAgA(cute::_, cute::_, cute::_, itile), tArA);
      cute::copy(tBgB(cute::_, cute::_, cute::_, itile), tBrB);

      cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
    }

    if (use_relu) {
      // 手动遍历并应用 ReLU
      // cute::size(tCrC) 是 8 = （kTileM * kTileN） / threadIdx.x
      // 所以应该就表示了 当前 threadIdx.x 要处理的元素
      CUTE_UNROLL
      for (int i = 0; i < cute::size(tCrC); ++i) {
          T2 val = tCrC(i);  // 取出当前值
          tCrC(i) = (val > T2(0)) ? val : T2(0);  // 应用 ReLU 操作
      }

      __syncthreads();
    }

    cute::copy(tCrC, tCgC);
}

template <typename input_dtype, typename output_dtype, bool use_relu, std::enable_if_t<!std::is_same_v<input_dtype, Half>, int> =0>
void _extended_gemm_kernel_low_level_api(
  input_dtype * a_ptr,
  input_dtype * b_ptr,
  output_dtype * out_ptr,
  int M,
  int N,
  int K,
  int lda,
  int ldb,
  int ldc) {
  TORCH_CHECK(false, "None Half input not support yet");
}


template <typename input_dtype, typename output_dtype, bool use_relu, std::enable_if_t<std::is_same_v<input_dtype, Half>, int> =0> // std::enable_if_t<std::is_same_v<input_dtype, Half>, int> =0
void _extended_gemm_kernel_low_level_api(
  input_dtype * a_ptr,
  input_dtype * b_ptr,
  output_dtype * out_ptr,
  int M,
  int N,
  int K,
  int lda,
  int ldb,
  int ldc) {
  constexpr int kTileM = 32;
  constexpr int kTileN = 32;
  constexpr int kTileK = 32;
  int grid_m = M / kTileM;
  int grid_n = N / kTileN;

  using mma_op = cute::SM80_16x8x16_F16F16F16F16_TN;
  using mma_traits = cute::MMA_Traits<mma_op>;
  using mma_atom = cute::MMA_Atom<mma_traits>;

  // mma_op will calculate MNK as 16x8x16，tensor core 作用在warp上，所以是一个warp的32线程 的计算size
  // 创建 tiled mma 通过 make_layout 可以进一步扩展  tiled_mma 计算size
  // param: thr layout 表示可以有更多的 warp(线程 来参与 mma计算)， 这里就是 M 维度是2个，N 维度是2个
  // param: val layout (**重要** 最新的code 下面 val layout 的参数被移除了) 表示 在 warp 内部，可以重复参与的mma 计算，这里表示 沿着 N 维度重复2次
  // param: 最后一个参数是 Permutations 被保留了： 
  //   refer to: https://github.com/NVIDIA/cutlass/blob/5e497243f7ad13a2aa842143f9b10bbb23d98292/include/cute/atom/mma_atom.hpp#L207
  //   refer to: https://zhuanlan.zhihu.com/p/28168438940
  // 综上：
  // 一共有 4 个 warp，128个线程参与计算, size(MMA{}) 返回的也是这个结构 (32, cute::_2, cute::_2, cute::_1)
  using MMA = decltype(make_tiled_mma(mma_atom{}, 
                      make_layout(cute::Shape<cute::_2, cute::_2, cute::_1>{}))); // thr layout
                      // make_layout(cute::Shape<cute::_1, cute::_2, cute::_1>{}))); // val layout has been removed


  using mma_op_fp32 = cute::SM80_16x8x16_F32F16F16F32_TN;
  using mma_traits_fp32 = cute::MMA_Traits<mma_op_fp32>;
  using mma_atom_fp32 = cute::MMA_Atom<mma_traits_fp32>;
  using MMA_fp32 = decltype(make_tiled_mma(mma_atom_fp32{}, 
                      make_layout(cute::Shape<cute::_2, cute::_2, cute::_1>{}))); // thr layout
                      // make_layout(cute::Shape<cute::_1, cute::_2, cute::_1>{}))); // val layout has been removed

  dim3 grid(grid_n, grid_m);
  dim3 block;
  using T = cute::half_t;
  using T2 = float;
  if constexpr (std::is_same_v<output_dtype, Half>) {
    block = dim3(size(MMA{}));
    _extended_gemm_block_cutlass_naive_kernel<T, T, kTileM, kTileN, kTileK, MMA, use_relu><<<grid, block>>>(
      (T*)a_ptr, (T*)b_ptr, (T*)out_ptr, M, N, K, lda, ldb, ldc
    );
  } else {
    block = dim3(size(MMA_fp32{}));
    _extended_gemm_block_cutlass_naive_kernel<T, T2, kTileM, kTileN, kTileK, MMA_fp32, use_relu><<<grid, block>>>(
      (T*)a_ptr, (T*)b_ptr, (T2*)out_ptr, M, N, K, lda, ldb, ldc
    );
  }

}

Tensor extended_gemm_kernel(
  Tensor a,
  Tensor b,
  Tensor out,
  std::string_view epilogue,
  bool transpose_B,
  int64_t api_level) {
    if (epilogue == "none" && !transpose_B && api_level == 0) {
      // High level API not support epilogue
      // TODO<leslie> assert the scalar_type is float, and the input tensor is 2D
      auto a_ptr = a.data_ptr();
      auto b_ptr = b.data_ptr();
      auto out_ptr = out.data_ptr();

      int M = a.size(0);
      int K = a.size(1);
      int N = b.size(1);

      int lda = a.size(1);
      int ldb = b.size(1);
      int ldc = out.size(1);

      _extended_gemm_kernel((float*)a_ptr, (float*)b_ptr, (float*)out_ptr, M, N, K, lda, ldb, ldc);
    } else if (api_level == 1) {
      // Collective API
      int M = a.size(0);
      int K = a.size(1);
      int N = b.size(0);

      int lda = a.size(1);
      int ldb = b.size(1);
      int ldc = out.size(1);
      // std::cout<<std::is_same_v<c10::impl::ScalarTypeToCPPTypeT<at::ScalarType::Half>, Half><<std::endl;
      AT_DISPATCH_FLOATING_TYPES_AND2(
          at::ScalarType::BFloat16, at::ScalarType::Half, out.scalar_type(),
          "_extended_gemm_kernel_collective_api_kernel_impl",
          [&] { 
            // std::cout<<std::is_same_v<scalar_t, Half><<std::endl;
            Half* a_ptr = a.data_ptr<Half>();
            Half* b_ptr = b.data_ptr<Half>();
            scalar_t* out_ptr = out.data_ptr<scalar_t>();
            if (epilogue == "relu") {
              _extended_gemm_kernel_collective_api<Half, scalar_t, true>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            } else {
              _extended_gemm_kernel_collective_api<Half, scalar_t, false>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            }
          });
    
    }
    else {
      // TODO <leslie>: assert transpose_B is True
      // A is: M x K
      // B is: N x K
      // C is: M x N

      int M = a.size(0);
      int K = a.size(1);
      int N = b.size(0);

      int lda = a.size(1);
      int ldb = b.size(1);
      int ldc = out.size(1);
      // std::cout<<std::is_same_v<c10::impl::ScalarTypeToCPPTypeT<at::ScalarType::Half>, Half><<std::endl;
      AT_DISPATCH_FLOATING_TYPES_AND2(
          at::ScalarType::BFloat16, at::ScalarType::Half, out.scalar_type(),
          "_extended_gemm_kernel_low_level_api_kernel_impl",
          [&] { 
            // std::cout<<std::is_same_v<scalar_t, Half><<std::endl;
            Half* a_ptr = a.data_ptr<Half>();
            Half* b_ptr = b.data_ptr<Half>();
            scalar_t* out_ptr = out.data_ptr<scalar_t>();
            if (epilogue == "relu") {
              _extended_gemm_kernel_low_level_api<Half, scalar_t, true>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            } else {
              _extended_gemm_kernel_low_level_api<Half, scalar_t, false>(a_ptr, b_ptr, out_ptr, M, N, K, lda, ldb, ldc);
            }
          });
    }

    return out;
}

} // namespace native
} // namespace at
