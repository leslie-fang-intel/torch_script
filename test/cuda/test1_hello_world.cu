// Build: /usr/local/cuda-12.1/bin/nvcc test_hello_world.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
    printf("Hello World. \n");
}

int main() {
    dkernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
