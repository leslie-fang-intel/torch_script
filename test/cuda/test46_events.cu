// Build: /usr/local/cuda-12.1/bin/nvcc test_hello_world.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1

__global__ void dkernel(int* d) {

    printf("dkernel thread is: %d \n", *d);

}

__global__ void dkernel2() {

    printf("dkernel2 thread is: %d \n", threadIdx.x);

}

int main() {

    int* da;
    int* db;
    hipMalloc(&da, N * sizeof(int));
    hipMemset(da, 0, N * sizeof(int));
    hipMemset(db, 1, N * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    // Event 可以实现 stream 之间的通信
    dkernel<<<1, N, 0, s1>>>(da);
    hipEventRecord(start, s1);
    dkernel<<<1, N, 0, s1>>>(db);

    // s2 等待 event 执行之后才会执行
    hipStreamWaitEvent(s2, start, 0);
    dkernel2<<<1, N, 0, s2>>>();

    hipDeviceSynchronize();

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
